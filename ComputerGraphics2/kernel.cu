#include "hip/hip_runtime.h"
﻿#define _CRT_SECURE_NO_WARNINGS
#include <iostream>
#include <fstream>
#include <math.h>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

double* setMatrix(double* M, int len) {
    for (int i = 0; i < len; i++)
        for (int j = 0; j < len; j++)
            M[i * len + j] = 1 + ((int)(rand() % 120));
    return M;
}

void printMatrix(double* M, int len) {
    for (int i = 0; i < len; i++) {
        for (int j = 0; j < len; j++)
            printf("%lf\t", M[i * len + j]);
        printf("\n");
    }printf("-----------------\n");
}

__device__ void kof(double* M, int i, int j, double tmp, int len) {
    int k = threadIdx.x;
    M[j * len + k] -= ((M[i * len + k] / M[i * len + i]) * tmp);
}


__global__ void gpuDet(double* M, int len) {
    if (len > 1) {
        for (int i = 0; i < len - 1; i++)
            for (int j = i + 1; j < len; j++) {
                double tmp = M[j * len + i];
                kof(M, i, j, tmp, len);
            }
    }
}


double det(double* M, int len) {

    int MatrixSize = len * len;
    int byteSize = MatrixSize * sizeof(double);
    double* inMatrix_d;
    hipMalloc((void**)&inMatrix_d, byteSize);
    hipMemcpy(inMatrix_d, M, byteSize, hipMemcpyHostToDevice);

    gpuDet <<< 1, len >>> (inMatrix_d, len);
    hipDeviceSynchronize();

    hipMemcpy(M, inMatrix_d, byteSize, hipMemcpyDeviceToHost);
    hipFree(inMatrix_d);

    double det = M[0 * len + 0];
    for (int i = 1; i < len; i++)
        det *= M[i * len + i];
    return det;
}

void laba2() {
    int lenArr[12] = { 2,4,8,16,32,64,128,256,350,512,738,1024 };
    int iterations = 12;
    double* Matrix;
    int len;
    ofstream f;
    f.open("resultsGPU.txt");
    if (f.is_open()) {
        f << "len: \tTime:\n";

        for (int k = 0; k < iterations; k++) {

            len = lenArr[k];

            Matrix = new double[len * len];
            Matrix = setMatrix(Matrix, len);

            //printMatrix(Matrix, len);
            auto start1 = chrono::high_resolution_clock::now();
            det(Matrix, len);
            auto end1 = chrono::high_resolution_clock::now();



            delete[] Matrix;

            double time_taken1 = chrono::duration_cast<chrono::nanoseconds>(end1 - start1).count();
            time_taken1 *= 1e-9;

            cout << len << " \t" << time_taken1 << endl;
			f << len << " \t" << time_taken1 << endl;
        }
    }
}

void main() {
    laba2();
}